#include "agdsgpu.cuh"

#include <hipsparse.h>
#include <hipblas.h>
#include "hip/hip_runtime.h"
#include ""

#include <iostream>


void CONN::addON(int vnOffset) {
	cooNzValuesDev.append(1.0);
	cooRowIndicesDev.append(nOn);
	cooColIndicesDev.append(vnOffset);

	nOn++;
	if (vnOffset == nVn) {
		nVn++;
	}

	freeDescr();
}


hipsparseSpMatDescr_t CONN::getDescr() {
	if (!isDescrReady) {
		CHECK_CUSPARSE(hipsparseCreateCoo(
			&descr,
			nOn,
			nVn,
			nOn,
			cooRowIndicesDev.valuesDev,
			cooColIndicesDev.valuesDev,
			cooNzValuesDev.valuesDev,
			HIPSPARSE_INDEX_32I,
			HIPSPARSE_INDEX_BASE_ZERO,
			HIP_R_32F
		));

		isDescrReady = true;
	}

	return descr;
}


void CONN::freeDescr() {
	if (isDescrReady) {
		CHECK_CUSPARSE(hipsparseDestroySpMat(descr));
		isDescrReady = false;
	}
}


CONN::CONN() {
	CHECK_CUDA(hipStreamCreate(&stream));
}


CONN::~CONN() {
	freeDescr();
	CHECK_CUDA(hipStreamDestroy(stream));
}
