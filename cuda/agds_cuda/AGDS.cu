#include "hip/hip_runtime.h"
#include "agdsgpu.cuh"

#include <hipsparse.h>
#include <hipblas.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

#include "measurements.hpp"


__constant__ const float alphabeta = 1.0;


AGDS::AGDS(float* data, int n_on, int n_vng) {
	CHECK_CUSPARSE(hipsparseCreate(&cusparseHandle));
	CHECK_CUBLAS(hipblasCreate(&cublasHandle));

	this->n_vng = n_vng;
	vngs.resize(n_vng);

	float* observation = new float[n_vng];

	for (int on_ix = 0; on_ix < n_on; on_ix++) {
		for (int vng_ix = 0; vng_ix < n_vng; vng_ix++) {
			observation[vng_ix] = data[vng_ix * n_on + on_ix];
		}

		addObservation(observation);
	}

	delete[] observation;
}

AGDS::~AGDS() {
	// freeAOnDescr();

	CHECK_CUSPARSE(hipsparseDestroy(cusparseHandle));
	CHECK_CUBLAS(hipblasDestroy(cublasHandle));
}


void AGDS::addObservation(float* values) {
	// int mesId = measurer.startMeasurement();

	for (int vng_ix = 0; vng_ix < n_vng; vng_ix++) {
		vngs[vng_ix].addValue(values[vng_ix]);
	}

	n_on++;

	freeAOnDescr();

	// measurer.endMeasurement(mesId);

	/*if (n_on % 1000 == 0) {
		std::cout << n_on << ", " << measurer.getElapsedTimeInSeconds(mesId) << std::endl;
	}*/
}


void AGDS::setupOnQuery(int* activatedOns, int nActivatedOns) {
	if (!isAOnReady) {
		AOn = new float[n_vn()];
		CHECK_CUDA(hipMalloc(&AOnDev, n_on * sizeof(float)));
		CHECK_CUSPARSE(hipsparseCreateDnVec(&AOnDescr, n_on, AOnDev, HIP_R_32F));
		isAOnReady = true;
	}

	const float initialActivation = 1.0;

	CHECK_CUDA(hipMemset(AOnDev, 0, n_on * sizeof(float)));
	for (int act_on_ix = 0; act_on_ix < nActivatedOns; act_on_ix++) {
		CHECK_CUDA(hipMemcpy(AOnDev + activatedOns[act_on_ix], &initialActivation, sizeof(float), hipMemcpyHostToDevice));
	}

	for (auto&& vng : vngs) {
		vng.resetAVn();
	}
}


void AGDS::freeAOnDescr() {
	if (isAOnReady) {
		CHECK_CUSPARSE(hipsparseDestroyDnVec(AOnDescr));
		delete[] AOn;
		CHECK_CUDA(hipFree(AOnDev));
		isAOnReady = false;
	}
}


int AGDS::n_vn() {
	int n_vn = 0;
	for (auto&& vng : vngs) {
		n_vn += vng.getNVn();
	}
	return n_vn;
}


void AGDS::printAOn() {
	CHECK_CUDA(hipMemcpy(AOn, AOnDev, n_on * sizeof(float), hipMemcpyDeviceToHost));
	std::cout << "AOn: ";

	for (int i = 0; i < n_on; i++) {
		std::cout << AOn[i] << " ";
	}
	std::cout << std::endl;
}


void AGDS::loadResultsToHost() {
	for (auto&& vng : vngs) {
		vng.loadAVnToHost();
	}

	CHECK_CUDA(hipMemcpy(AOn, AOnDev, n_on * sizeof(float), hipMemcpyDeviceToHost));
}


void AGDS::infere() {
	/*for (auto& vng : vngs) {
		CHECK_CUDA(hipEventCreate(&vng.vngReadyForVn2OnEvent));
	}*/
	
	on2vn();
	vn2vn();
	vn2on();

	loadResultsToHost();

	/*for (auto& vng : vngs) {
		CHECK_CUDA(hipEventDestroy(vng.vngReadyForVn2OnEvent));
	}*/
}


void AGDS::on2vn() {
	hipsparseOperation_t opTranspose = HIPSPARSE_OPERATION_TRANSPOSE;
	hipsparseSpMVAlg_t algorithm = HIPSPARSE_SPMV_COO_ALG1;
	hipDataType valueType = HIP_R_32F;

	for (auto&& vng : vngs) {
		void* buffer;
		size_t bufferSize;

		// hipStream_t on2vnStream = vng.stream;
		hipStream_t on2vnStream = 0;

		// CHECK_CUSPARSE(hipsparseSetStream(cusparseHandle, on2vnStream));
		CHECK_CUSPARSE(
			hipsparseSpMV_bufferSize(
				cusparseHandle,
				opTranspose,
				&alphabeta,
				vng.Conn.getDescr(),
				AOnDescr,
				&alphabeta,
				vng.getAVnDescr(),
				valueType,
				algorithm,
				&bufferSize
			)
		);

		CHECK_CUDA(hipMallocAsync(&buffer, bufferSize, on2vnStream));

		/*vng.Conn.cooNzValuesDev.print("COO vals");
		vng.Conn.cooRowIndicesDev.print("COO rows");
		vng.Conn.cooColIndicesDev.print("COO cols");

		printAOn();
		vng.printAVn();*/

		// CHECK_CUSPARSE(hipsparseSetStream(cusparseHandle, on2vnStream));
		CHECK_CUSPARSE(hipsparseSpMV(
			cusparseHandle,
			opTranspose,
			&alphabeta,
			vng.Conn.getDescr(),
			AOnDescr,
			&alphabeta,
			vng.getAVnDescr(),
			valueType,
			algorithm,
			buffer
		));

		CHECK_CUDA(hipFreeAsync(buffer, on2vnStream));

		/*printAOn();
		vng.printAVn();*/
	}
}


__global__ void computeWeights(float* prodSingleVng, float* weights, int vngSize) {
	int targetVnIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int sourceVnIdx = blockIdx.y * blockDim.y + threadIdx.y;

	if (targetVnIdx < vngSize && sourceVnIdx < vngSize) {
		weights[sourceVnIdx * vngSize + targetVnIdx] =
			targetVnIdx > sourceVnIdx ?
			prodSingleVng[targetVnIdx] / prodSingleVng[sourceVnIdx]
			: prodSingleVng[sourceVnIdx] / prodSingleVng[targetVnIdx];
	}
}

void AGDS::vn2vn() {
	for (auto&& vng : vngs) {
		int vngSize = vng.getNVn();

		// hipStream_t vn2vnStream = vng.stream;
		hipStream_t vn2vnStream = 0;

		float* weightsMatDev;
		CHECK_CUDA(hipMallocAsync(&weightsMatDev, vngSize * vngSize * sizeof(float), vn2vnStream));

		dim3 threadsPerBlock(16, 16);
		dim3 nBlocks((vngSize + threadsPerBlock.x - 1) / threadsPerBlock.x, (vngSize + threadsPerBlock.y - 1) / threadsPerBlock.y);
		computeWeights <<<nBlocks, threadsPerBlock, 0, vn2vnStream >>> (vng.P->valuesDev, weightsMatDev, vngSize);

		// CHECK_CUBLAS(hipblasSetStream(cublasHandle, vn2vnStream));
		CHECK_CUBLAS(hipblasSgemv(
			cublasHandle,
			HIPBLAS_OP_N,
			vngSize,
			vngSize,
			&alphabeta,
			weightsMatDev,
			vngSize,
			vng.AVnDev,
			1,
			&alphabeta,
			vng.AVnDev,
			1
		));

		// CHECK_CUDA(hipEventRecord(vng.vngReadyForVn2OnEvent, vn2vnStream))
		CHECK_CUDA(hipFreeAsync(weightsMatDev, vn2vnStream));
	}
}


__global__ void VecMulElementwise(float* a, float* b, float* res, int len) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len) {
		res[i] = a[i] * b[i];
	}
}

void AGDS::vn2on() {
	hipsparseOperation_t opTranspose = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	const float alphabeta = 1.0;
	hipsparseSpMVAlg_t algorithm = HIPSPARSE_SPMV_COO_ALG1;
	hipDataType valueType = HIP_R_32F;

	const int blockSize = 512;

	hipStream_t mainStream = 0;
	// 	CHECK_CUDA(hipStreamCreate(&mainStream));

	for (auto&& vng : vngs) {
		// hipStream_t on2vnStream = vng.stream;
		hipStream_t on2vnStream = 0;

		float* weightedAVnDev;
		CHECK_CUDA(hipMallocAsync(&weightedAVnDev, vng.getNVn() * sizeof(float), on2vnStream));

		int nBlocks = vng.getNVn() / blockSize + 1;
		VecMulElementwise << < nBlocks, blockSize, 0, on2vnStream >> > (vng.AVnDev, vng.Nrev->valuesDev, weightedAVnDev, vng.getNVn());

		hipsparseDnVecDescr_t weightedAVnDescr;
		// CHECK_CUSPARSE(hipsparseSetStream(cusparseHandle, on2vnStream));
		CHECK_CUSPARSE(hipsparseCreateDnVec(&weightedAVnDescr, vng.getNVn(), weightedAVnDev, HIP_R_32F));

		void* buffer;
		size_t bufferSize;

		// CHECK_CUSPARSE(hipsparseSetStream(cusparseHandle, on2vnStream));
		CHECK_CUSPARSE(
			hipsparseSpMV_bufferSize(
				cusparseHandle,
				opTranspose,
				&alphabeta,
				vng.Conn.getDescr(),
				weightedAVnDescr,
				&alphabeta,
				AOnDescr,
				valueType,
				algorithm,
				&bufferSize
			)
		);

		CHECK_CUDA(hipMallocAsync(&buffer, bufferSize, on2vnStream));

		// CHECK_CUDA(hipStreamWaitEvent(mainStream, vng.vngReadyForVn2OnEvent));
		// CHECK_CUSPARSE(hipsparseSetStream(cusparseHandle, mainStream));
		CHECK_CUSPARSE(hipsparseSpMV(
			cusparseHandle,
			opTranspose,
			&alphabeta,
			vng.Conn.getDescr(),
			weightedAVnDescr,
			&alphabeta,
			AOnDescr,
			valueType,
			algorithm,
			buffer
		));

		CHECK_CUDA(hipFreeAsync(buffer, mainStream));
		
		// CHECK_CUSPARSE(hipsparseSetStream(cusparseHandle, mainStream));
		CHECK_CUSPARSE(hipsparseDestroyDnVec(weightedAVnDescr));
		CHECK_CUDA(hipFreeAsync(weightedAVnDev, mainStream));
	}

	// CHECK_CUDA(hipStreamDestroy(mainStream));
}
